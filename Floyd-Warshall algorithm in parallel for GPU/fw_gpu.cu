#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include "workshop.h"

#define GRAPH_SIZE 2000
#define THREADS_PER_BLOCK 1024
#define BLOCKS MIN(32, (GRAPH_SIZE + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK)
#define TILE_WIDTH 32
#define TILE_HEIGHT 32


#define EDGE_COST(graph, graph_size, a, b) graph[a * graph_size + b]
#define D(a, b) EDGE_COST(output, graph_size, a, b)

#define INF 0x1fffffff

__device__
int Min(int a, int b) { return a < b ? a : b; }

void generate_random_graph(int *output, int graph_size) {
  int i, j;

  srand(0xdadadada);

  for (i = 0; i < graph_size; i++) {
    for (j = 0; j < graph_size; j++) {
      if (i == j) {
        D(i, j) = 0;
      } else {
        int r;
        r = rand() % 40;
        if (r > 20) {
          r = INF;
        }

        D(i, j) = r;
      }
    }
  }
}

__global__ void floyd_warshall_gpu(int graph_size, int *output, int k) {
  int i, j;
  i = blockIdx.x * blockDim.x + threadIdx.x; 
  j = blockIdx.y * blockDim.y + threadIdx.y;
  while(i < graph_size && j < graph_size){
      if (D(i, k) + D(k, j) < D(i, j)) {
        D(i , j) = D(i, k) + D(k, j);
        __syncthreads();
    }
  }
}

void floyd_warshall_cpu(const int *graph, int graph_size, int *output) {
  int i, j, k;

  memcpy(output, graph, sizeof(int) * graph_size * graph_size);

  for (k = 0; k < graph_size; k++) {
    for (i = 0; i < graph_size; i++) {
      for (j = 0; j < graph_size; j++) {
        if (D(i, k) + D(k, j) < D(i, j)) {
          D(i , j) = D(i, k) + D(k, j);
        }
      }
    }
  }
}

int main(int argc, char **argv) {
  #define TIMER_START() gettimeofday(&tv1, NULL)
  #define TIMER_STOP()                                                           \
    gettimeofday(&tv2, NULL);                                                    \
    timersub(&tv2, &tv1, &tv);                                                   \
    time_delta = (float)tv.tv_sec + tv.tv_usec / 1000000.0
  
    struct timeval tv1, tv2, tv;
    float time_delta;
  
    int *graph, *output_cpu, *host_output_gpu, *output_gpu;
    int size;
  
    size = sizeof(int) * GRAPH_SIZE * GRAPH_SIZE;
  
    graph = (int *)malloc(size);
    assert(graph);
  
    host_output_gpu = (int *)malloc(size);
    assert(host_output_gpu);
    memset(host_output_gpu, 0, size);

    output_cpu = (int *)malloc(size);
    assert(output_cpu);
    memset(output_cpu, 0, size);
  
    output_gpu = (int *)malloc(size);
    assert(output_gpu);

    generate_random_graph(graph, GRAPH_SIZE);
  
    fprintf(stderr, "running on cpu...\n");
    TIMER_START();
    floyd_warshall_cpu(graph, GRAPH_SIZE, output_cpu);
    TIMER_STOP();
    fprintf(stderr, "%f secs\n", time_delta);
    
    HANDLE_ERROR(hipMalloc(&output_gpu, size));
    hipMemcpy(output_gpu, graph, size, hipMemcpyHostToDevice);

    fprintf(stderr, "running on gpu...\n");
    TIMER_START();
    for (int k = 0; k < GRAPH_SIZE; k++) {
      floyd_warshall_gpu<<<BLOCKS, THREADS_PER_BLOCK>>>(GRAPH_SIZE, output_gpu, k);
    }
    TIMER_STOP();
    
    hipMemcpy(graph, output_gpu, size, hipMemcpyDeviceToHost);

    fprintf(stderr, "%f secs\n", time_delta);

    if (memcmp(output_cpu, host_output_gpu, size) != 0) {
      fprintf(stderr, "FAIL!\n");
    }

    hipFree(output_gpu);


    free(graph);
    free(output_cpu);
    free(host_output_gpu);
    
  
    return 0;
  }